#include "hip/hip_runtime.h"
#include "../../include/cuda/cuda_utils.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Declare the CUDA kernels
__global__ void attention_scores_kernel(const float *Q, const float *K,
                                        float *scores, const float scale,
                                        int seq_len, int head_dim) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < seq_len && col < seq_len) {
    float sum = 0.0f;
    for (int i = 0; i < head_dim; i++) {
      sum += Q[row * head_dim + i] * K[col * head_dim + i];
    }
    scores[row * seq_len + col] = sum * scale;
  }
}

__global__ void softmax_kernel(float *scores, int seq_len) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < seq_len) {
    float max_val = scores[row * seq_len];
    for (int i = 1; i < seq_len; i++) {
      max_val = max(max_val, scores[row * seq_len + i]);
    }

    float sum = 0.0f;
    for (int i = 0; i < seq_len; i++) {
      scores[row * seq_len + i] = expf(scores[row * seq_len + i] - max_val);
      sum += scores[row * seq_len + i];
    }

    for (int i = 0; i < seq_len; i++) {
      scores[row * seq_len + i] /= sum;
    }
  }
}

// CUDA kernel launcher without template
void launch_attention_scores_kernel(const float *Q, const float *K,
                                    float *scores, float scale, int seq_len,
                                    int head_dim, hipStream_t stream) {
  dim3 block_dim(16, 16);
  dim3 grid_dim((seq_len + block_dim.x - 1) / block_dim.x,
                (seq_len + block_dim.y - 1) / block_dim.y);

  attention_scores_kernel<<<grid_dim, block_dim, 0, stream>>>(
      Q, K, scores, scale, seq_len, head_dim);
}

void launch_softmax_kernel(float *scores, int seq_len, hipStream_t stream) {
  dim3 block_dim(256);
  dim3 grid_dim((seq_len + block_dim.x - 1) / block_dim.x);

  softmax_kernel<<<grid_dim, block_dim, 0, stream>>>(scores, seq_len);
}

Matrix cuda_matmul(const Matrix &A, const Matrix &B) {
  std::cout << "Starting CUDA matrix multiplication..." << std::endl;
  std::cout << "Matrix A: " << A.rows() << "x" << A.cols() << std::endl;
  std::cout << "Matrix B: " << B.rows() << "x" << B.cols() << std::endl;

  hipblasHandle_t handle;
  hipblasStatus_t status;
  hipError_t err;

  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error("Failed to create cuBLAS handle");
  }

  float alpha = 1.0f;
  float beta = 0.0f;

  Matrix C(A.rows(), B.cols(), 0.0f);
  Matrix C_gpu = C.to_gpu();

  try {
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, B.cols(), A.rows(),
                         A.cols(), &alpha, B.get_data(), B.cols(), A.get_data(),
                         A.cols(), &beta, C_gpu.get_data(), C_gpu.cols());

    if (status != HIPBLAS_STATUS_SUCCESS) {
      throw std::runtime_error("cuBLAS SGEMM failed with status: " +
                               std::to_string(status));
    }

    // Synchronize to catch any asynchronous errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      throw std::runtime_error("CUDA sync failed: " +
                               std::string(hipGetErrorString(err)));
    }

    std::cout << "CUDA matrix multiplication completed successfully"
              << std::endl;
    C = C_gpu.to_cpu();
  } catch (const std::exception &e) {
    hipblasDestroy(handle);
    throw;
  }

  hipblasDestroy(handle);
  return C;
}