#include "hip/hip_runtime.h"
#define USE_CUDA
#include "../../include/cuda/cuda_check.cuh"
#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/layer_norm.hpp"
#include "cuda/layernorm_kernels.cuh"
#include "../../include/cuda/backward_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

#ifdef USE_CUDA

namespace cuda {

__global__ void LayerNormBackwardKernel(
    const float* d_grad_output,
    const float* d_input,
    const float* d_gamma,
    float* d_grad_gamma,
    const int batch_size,
    const int hidden_size,
    const float eps
) {
    // ... kernel implementation ...
}

void LayerNormBackwardCUDA(
    const float* d_grad_output,
    const float* d_input,
    const float* d_gamma,
    float* d_grad_gamma,
    const int batch_size,
    const int hidden_size,
    const float eps
) {
    // Calculate grid and block dimensions
    dim3 block(256);
    dim3 grid((batch_size * hidden_size + block.x - 1) / block.x);

    // Launch kernel with correct parameter types
    LayerNormBackwardKernel<<<grid, block>>>(
        d_grad_output,
        d_input,
        d_gamma,
        d_grad_gamma,
        batch_size,
        hidden_size,
        eps
    );
}

void layer_norm_backward(const Matrix& grad_output, const Matrix& input,
                         const Matrix& gamma, Matrix& grad_gamma,
                         Matrix& grad_beta, float eps) {
    std::cout << "\n=== LayerNorm Backward Debug ===" << std::endl << std::flush;
    std::cout << "grad_output dims: " << grad_output.rows() << "x" << grad_output.cols() << std::endl << std::flush;
    std::cout << "input dims: " << input.rows() << "x" << input.cols() << std::endl << std::flush;
    std::cout << "gamma size: " << gamma.size() << std::endl;
    std::cout << "grad_gamma size: " << grad_gamma.size() << std::endl;
    std::cout << "grad_beta size: " << grad_beta.size() << std::endl;

    int batch_size = input.rows();
    int hidden_size = input.cols();

    std::cout << "Allocating device memory..." << std::endl;
    // Allocate device memory
    float *d_grad_output, *d_input, *d_gamma;
    float *d_grad_gamma, *d_grad_beta;
    std::cout << "Allocating device memory..." << std::endl;
    std::cout << "batch_size: " << batch_size << ", hidden_size: " << hidden_size << std::endl;
    std::cout << "grad_output size: " << grad_output.size() << std::endl;
    std::cout << "input size: " << input.size() << std::endl;
    std::cout << "gamma size: " << gamma.size() << std::endl;
    std::cout << "grad_gamma size: " << grad_gamma.size() << std::endl;
    std::cout << "grad_beta size: " << grad_beta.size() << std::endl;
    CUDA_CHECK(hipMalloc(&d_grad_output, batch_size * hidden_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_input, batch_size * hidden_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gamma, hidden_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_gamma, hidden_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad_beta, hidden_size * sizeof(float)));

    std::cout << "Copying data to device..." << std::endl;
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_grad_output, grad_output.data(), 
                        batch_size * hidden_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_input, input.data(), 
                        batch_size * hidden_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_gamma, gamma.data(), 
                        hidden_size * sizeof(float), hipMemcpyHostToDevice));

    std::cout << "Launching kernel..." << std::endl;
    // Launch kernel
    // Use smaller block size to ensure enough blocks for all features
    dim3 block(32, 1);  // 32 threads per block is more reasonable
    // Calculate grid size to cover all features
    int num_blocks = (hidden_size + block.x - 1) / block.x;
    dim3 grid(num_blocks, 1);

    // Calculate shared memory size needed for mean and variance
    // Each block needs space for its own mean and variance arrays
    size_t shared_mem_size = 2 * block.x * sizeof(float);  // 2 arrays of 32 floats each
    
    // Verify shared memory size is sufficient
    size_t max_shared_mem = 48 * 1024;  // 48KB typical limit
    if (shared_mem_size > max_shared_mem) {
        printf("Error: Required shared memory (%zu bytes) exceeds maximum (%zu bytes)\n", 
               shared_mem_size, max_shared_mem);
        return;
    }
    
    // Zero out grad arrays before kernel launch
    CUDA_CHECK(hipMemset(d_grad_gamma, 0, hidden_size * sizeof(float)));
    CUDA_CHECK(hipMemset(d_grad_beta, 0, hidden_size * sizeof(float)));

    LayerNormBackwardKernel<<<grid, block, shared_mem_size>>>(
        d_grad_output, d_input, d_gamma, d_grad_gamma, batch_size, hidden_size, eps);
    
    // Ensure kernel completion before proceeding
    CUDA_CHECK(hipDeviceSynchronize());

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
    }

    std::cout << "Copying results back to host..." << std::endl;
    std::cout << "Copying grad_gamma..." << std::endl;
    CUDA_CHECK(hipMemcpy(grad_gamma.data(), d_grad_gamma, hidden_size * sizeof(float),
                        hipMemcpyDeviceToHost));
    std::cout << "Copying grad_beta..." << std::endl;
    CUDA_CHECK(hipMemcpy(grad_beta.data(), d_grad_beta, hidden_size * sizeof(float),
                        hipMemcpyDeviceToHost));

    std::cout << "Freeing device memory..." << std::endl;
    // Free device memory
    CUDA_CHECK(hipFree(d_grad_output));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_gamma));
    CUDA_CHECK(hipFree(d_grad_gamma));
    CUDA_CHECK(hipFree(d_grad_beta));
    std::cout << "=== LayerNorm Backward Complete ===" << std::endl;
}

void layer_norm_forward(const Matrix& input, const Matrix& gamma, const Matrix& beta,
                          Matrix& output, float eps) {
    const int batch_size = input.rows();
    const int hidden_size = input.cols();
    
    float* d_input, *d_gamma, *d_beta, *d_output;
    float* d_mean, *d_variance;
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_input, input.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gamma, gamma.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_beta, beta.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, output.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_mean, batch_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_variance, batch_size * sizeof(float)));
    
    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_gamma, gamma.data(), gamma.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.data(), beta.size() * sizeof(float), hipMemcpyHostToDevice));
    
    // Launch kernels
    const int block_size = 256;
    const int grid_size = (batch_size + block_size - 1) / block_size;
    
    // First compute mean and variance
    size_t shared_mem_size = 2 * block_size * sizeof(float);  // For sum and squared sum
    layer_norm_stats_kernel<<<grid_size, block_size, shared_mem_size>>>(
        d_input, d_mean, d_variance, hidden_size, batch_size);
    
    // Then normalize using the computed statistics
    const int total_elements = batch_size * hidden_size;
    const int norm_grid_size = (total_elements + block_size - 1) / block_size;
    layer_norm_kernel<<<norm_grid_size, block_size>>>(
        d_input, d_mean, d_variance, d_gamma, d_beta, d_output,
        hidden_size, batch_size, eps);
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost));
    
    // Free device memory
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_gamma));
    CUDA_CHECK(hipFree(d_beta));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_mean));
    CUDA_CHECK(hipFree(d_variance));
}

__global__ void layer_norm_stats_kernel(const float* input, float* mean, float* variance,
                                      int hidden_size, int batch_size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) return;

    float sum = 0.0f;
    float sq_sum = 0.0f;
    
    // Compute mean and variance for this sequence position
    for (int i = 0; i < hidden_size; ++i) {
        float val = input[idx * hidden_size + i];
        sum += val;
        sq_sum += val * val;
    }
    
    mean[idx] = sum / hidden_size;
    variance[idx] = (sq_sum / hidden_size) - (mean[idx] * mean[idx]);
}

__global__ void layer_norm_kernel(const float* input, const float* mean, const float* variance,
                                const float* gamma, const float* beta, float* output,
                                int hidden_size, int batch_size, float eps) {
    const int batch_idx = blockIdx.x;
    const int tid = threadIdx.x;
    
    if (batch_idx >= batch_size) return;
    
    // Load mean and variance for this sequence position
    const float mean_val = mean[batch_idx];
    const float var_val = variance[batch_idx];
    const float inv_std = rsqrtf(var_val + eps);
    
    // Normalize each element in the sequence
    for (int i = tid; i < hidden_size; i += blockDim.x) {
        const int idx = batch_idx * hidden_size + i;
        const float normalized = (input[idx] - mean_val) * inv_std;
        output[idx] = gamma[i] * normalized + beta[i];
    }
}

} // namespace cuda

#endif // USE_CUDA